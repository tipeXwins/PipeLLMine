--- qulacs/qulacs#196/after/update_ops_multi.cu	2022-01-10 16:02:54.000000000 +0000
+++ qulacs/qulacs#196/before/update_ops_multi.cu	2022-01-10 16:02:54.000000000 +0000
@@ -986,7 +986,7 @@
         if(target_qubit_index_count<=5){
 		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrix_const_gpu), matrix, sizeof(GTYPE)*matrix_dim*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrix_mask_list_gpu), matrix_mask_list, sizeof(ITYPE)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
-		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*(target_qubit_index_count + 1), 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
+		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 
             single_qubit_control_multi_qubit_dense_matrix_gate_const_gpu<<< grid, block, 0, *hip_stream >>> (control_qubit_index, control_value, target_qubit_index_count, state_gpu, dim);
         }else{
@@ -994,7 +994,7 @@
 		    checkCudaErrors(hipMemcpyAsync(d_matrix, matrix, matrix_dim *matrix_dim * sizeof(GTYPE), hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_matrix_mask_list), matrix_dim *matrix_dim * sizeof(GTYPE) ), __FILE__, __LINE__);
 		    checkCudaErrors(hipMemcpyAsync(d_matrix_mask_list, matrix_mask_list, sizeof(ITYPE)*matrix_dim, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
-		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*(target_qubit_index_count + 1), 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
+		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    
             single_qubit_control_multi_qubit_dense_matrix_gate_const_gpu<<< grid, block, 0, *hip_stream >>> (control_qubit_index, control_value, target_qubit_index_count, d_matrix, state_gpu, dim);
         }
@@ -1135,7 +1135,7 @@
         if(target_qubit_index_count<=5){
 		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrix_const_gpu), matrix, sizeof(GTYPE)*matrix_dim*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(matrix_mask_list_gpu), matrix_mask_list, sizeof(ITYPE)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
-		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*(target_qubit_index_count+control_qubit_index_count), 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
+		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 
             multi_qubit_control_multi_qubit_dense_matrix_gate_const_gpu<<< grid, block, 0, *hip_stream >>> (control_mask, target_qubit_index_count, control_qubit_index_count, state_gpu, dim);
         }else{
@@ -1143,7 +1143,7 @@
 		    checkCudaErrors(hipMemcpyAsync(d_matrix, matrix, matrix_dim *matrix_dim * sizeof(GTYPE), hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_matrix_mask_list), matrix_dim *matrix_dim * sizeof(GTYPE) ), __FILE__, __LINE__);
 		    checkCudaErrors(hipMemcpyAsync(d_matrix_mask_list, matrix_mask_list, sizeof(ITYPE)*matrix_dim, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
-		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*(target_qubit_index_count + control_qubit_index_count), 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
+		    checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(sorted_insert_index_list_gpu), sorted_insert_index_list, sizeof(UINT)*matrix_dim, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
 		    
             multi_qubit_control_multi_qubit_dense_matrix_gate_const_gpu<<< grid, block, 0, *hip_stream >>> (control_mask, target_qubit_index_count, control_qubit_index_count, d_matrix, state_gpu, dim);
         }
